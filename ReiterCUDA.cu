#include "hip/hip_runtime.h"
#include "ReiterCUDA.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <stdio.h>

__device__ void GetNeighbourCellIds(size_t cellId, size_t* outIdArray, int m_Width)
{
    size_t j = cellId % m_Width;
    size_t i = (cellId - j) / m_Width;

    int nOff;
    if (j%2 == 0)
        nOff = -1;
    else
        nOff = 0;
        
    outIdArray[0] = m_Width * (i-1) + j;
    outIdArray[1] = m_Width * (nOff + i) + j - 1;
    outIdArray[2] = m_Width * (nOff + i+1) + j - 1;
    outIdArray[3] = m_Width * (nOff + i) + j + 1;
    outIdArray[4] = m_Width * (nOff + i+1) + j + 1;
    outIdArray[5] = m_Width * (i+1) + j;
}

__device__ bool CheckReceptiveCell(float* data, size_t cellId, int m_Width, int m_Height)
{
    if(data[cellId] >= 1)
        return true;

    int j = cellId % m_Width;
    int i = (cellId - j) / m_Width;

    int nOff;
    if (j%2 == 0)
        nOff = -1;
    else
        nOff = 0;
        
    if(i>0 && data[m_Width * (i-1) + j] >= 1)
        return true;
    if(j>0 && (nOff + i) > 0 && data[m_Width * (nOff + i) + j - 1] >= 1)
        return true;
    if(j>0 && (nOff + i+1) < m_Height && data[m_Width * (nOff + i+1) + j - 1] >= 1)
        return true;
    if(j+1 < m_Width && (nOff + i) > 0 && data[m_Width * (nOff + i) + j + 1] >= 1)
        return true;
    if(j+1 < m_Width && (nOff + i+1) < m_Height && data[m_Width * (nOff + i+1) + j + 1] >= 1)
        return true;
    if(i+1 < m_Height && data[m_Width * (i+1) + j] >= 1)
        return true;

    return false;
}

__global__ void simulationKernel(float* curData, float* prevData, int height, int width, size_t* idArray, int maxIter, float alpha, float beta, float gamma)
{
    int cellId = blockIdx.x * blockDim.x + threadIdx.x;
    if (cellId < height * width)
    {
        int j = cellId % width;
        int i = (cellId - j) / width;

        if (i == 0 || j == 0 || height - i == 1 || width - j == 1)
            return;

        GetNeighbourCellIds(cellId, idArray, width);

        float sum = 0;
        for (int k = 0; k < 6; k++) {
            int id = idArray[k];
            if (!CheckReceptiveCell(prevData, id, width, height))
                sum += prevData[id];
        }

        float cellR = (CheckReceptiveCell(prevData, cellId, width, height) ? 1.0 : 0.0);
        float cellU = (cellR == 0.0 ? prevData[cellId] : 0.0);

        curData[cellId] = prevData[cellId] + (alpha / 2.0) * ((sum / 6.0) - cellU) + (gamma * cellR);
    }
}

double ReiterCUDA::RunSimulation(float alpha, float beta, float gamma)
{
    // Host data
    size_t* idArrayHost;

    // Device data
    float* curDataDevice;
    float* prevDataDevice;
    size_t* idArrayDevice;

    // Allocate host memory
    auto curDataHost = CreateGrid(beta);
    auto prevDataHost = CreateGrid(beta);
    idArrayHost = (size_t*)malloc(6 * sizeof(size_t));

    LogState(curDataHost.get(), 0);

    // Allocate device memory
    hipMalloc((void**)&curDataDevice, m_Height * m_Width * sizeof(float));
    hipMalloc((void**)&prevDataDevice, m_Height * m_Width * sizeof(float));
    hipMalloc((void**)&idArrayDevice, 6 * sizeof(size_t));

    auto start = std::chrono::high_resolution_clock::now();

    // Copy initial data to device
    hipMemcpy(curDataDevice, curDataHost.get(), m_Height * m_Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(prevDataDevice, prevDataHost.get(), m_Height * m_Width * sizeof(float), hipMemcpyHostToDevice);

    bool stable = false;
    size_t iter = 0;

    while (!stable && iter <= MAX_ITER)
    {
        stable = true;

        int blockSize = 256;
        int gridSize = (m_Height * m_Width + blockSize - 1) / blockSize;
        simulationKernel<<<gridSize, blockSize>>>(curDataDevice, prevDataDevice, m_Height, m_Width, idArrayDevice, MAX_ITER, alpha, beta, gamma);

        hipDeviceSynchronize();

        float* tmp;
        tmp = curDataDevice;
        curDataDevice = prevDataDevice;
        prevDataDevice = tmp;

        iter++;
        stable = false;
    }

    // Get data from device
    float* dataOutput = (float*)malloc(m_Height * m_Width * sizeof(float));
    hipMemcpy(dataOutput, curDataDevice, m_Height * m_Width * sizeof(float), hipMemcpyDeviceToHost);
    LogState(dataOutput, 1);

    auto stop = std::chrono::high_resolution_clock::now();
    printf("Simulation took %ld iterations\n", iter);
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    // Free device memory
    hipFree(curDataDevice);
    hipFree(prevDataDevice);
    hipFree(idArrayDevice);

    // Free host memory
    //free(curDataHost);
    //free(prevDataHost);
    free(idArrayHost);

    return (duration.count() * 1e-6);
}

int main(int argc, char** argv)
{
    ReiterCUDA model(100, 100);
    auto dur = model.RunSimulation(1, 0.2, 0.01);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Ran on %d GPU(s)\n", deviceCount);
    printf("Execution took %lf seconds\n", dur);

    return 0;
}
