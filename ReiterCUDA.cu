#include "hip/hip_runtime.h"
#include "ReiterCUDA.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <stdio.h>

__device__ void GetNeighbourCellIds(size_t cellId, size_t* outIdArray, int width)
{
    size_t j = cellId % width;
    size_t i = (cellId - j) / width;

    int nOff;
    if (j%2 == 0)
        nOff = -1;
    else
        nOff = 0;
        
    outIdArray[0] = width * (i-1) + j;
    outIdArray[1] = width * (nOff + i) + j - 1;
    outIdArray[2] = width * (nOff + i+1) + j - 1;
    outIdArray[3] = width * (nOff + i) + j + 1;
    outIdArray[4] = width * (nOff + i+1) + j + 1;
    outIdArray[5] = width * (i+1) + j;
}

__device__ bool CheckReceptiveCell(float* data, size_t cellId, int width, int height)
{
    if(data[cellId] >= 1)
        return true;

    int j = cellId % width;
    int i = (cellId - j) / width;

    int nOff;
    if (j%2 == 0)
        nOff = -1;
    else
        nOff = 0;
        
    if(i>0 && data[width * (i-1) + j] >= 1)
        return true;
    if(j>0 && (nOff + i) > 0 && data[width * (nOff + i) + j - 1] >= 1)
        return true;
    if(j>0 && (nOff + i+1) < height && data[width * (nOff + i+1) + j - 1] >= 1)
        return true;
    if(j+1 < width && (nOff + i) > 0 && data[width * (nOff + i) + j + 1] >= 1)
        return true;
    if(j+1 < width && (nOff + i+1) < height && data[width * (nOff + i+1) + j + 1] >= 1)
        return true;
    if(i+1 < height && data[width * (i+1) + j] >= 1)
        return true;

    return false;
}

__global__ void simulationKernel(float* curData, float* prevData, int height, int width, float alpha, float beta, float gamma)
{
    int cellId = blockIdx.x * blockDim.x + threadIdx.x;

    if (cellId >= height * width)
        return;

    size_t idArray[6];
    
    int j = cellId % width;
    int i = (cellId - j) / width;

    if (i == 0 || j == 0 || height - i == 1 || width - j == 1)
        return;

    GetNeighbourCellIds(cellId, idArray, width);

    float sum = 0;
    for (int k = 0; k < 6; k++) {
        int id = idArray[k];
        if (!CheckReceptiveCell(prevData, id, width, height))
            sum += prevData[id];
    }

    float cellR = (CheckReceptiveCell(prevData, cellId, width, height) ? 1.0 : 0.0);
    float cellU = (cellR == 0.0 ? prevData[cellId] : 0.0);

    curData[cellId] = prevData[cellId] + (alpha / 2.0) * ((sum / 6.0) - cellU) + (gamma * cellR);
    
}

double ReiterCUDA::RunSimulation(float alpha, float beta, float gamma)
{
    // Device data
    float* curDataDevice;
    float* prevDataDevice;

    // Allocate host memory
    auto hostGrid = CreateGrid(beta);

    // Allocate device memory
    hipMalloc((void**)&curDataDevice, m_Height * m_Width * sizeof(float));
    hipMalloc((void**)&prevDataDevice, m_Height * m_Width * sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();

    // Copy initial data to device
    hipMemcpy(curDataDevice, hostGrid.get(), m_Height * m_Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(prevDataDevice, hostGrid.get(), m_Height * m_Width * sizeof(float), hipMemcpyHostToDevice);

    size_t iter = 0;

    while (!IsStable(hostGrid.get()) && iter <= MAX_ITER)
    {
        int blockSize = 256;
        int gridSize = (m_Height * m_Width + blockSize - 1) / blockSize;
        simulationKernel<<<gridSize, blockSize>>>(curDataDevice, prevDataDevice, m_Height, m_Width, alpha, beta, gamma);

        hipDeviceSynchronize();

        auto tmp = curDataDevice;
        curDataDevice = prevDataDevice;
        prevDataDevice = tmp;

        hipMemcpy(hostGrid.get(), prevDataDevice, m_Height * m_Width * sizeof(float), hipMemcpyDeviceToHost);
        if(m_DebugFreq == DebugFreq::EveryIter)
            LogState(hostGrid.get(), iter);

        iter++;
    }

    // Get data from device
    hipMemcpy(hostGrid.get(), curDataDevice, m_Height * m_Width * sizeof(float), hipMemcpyDeviceToHost);
    if(m_DebugFreq == DebugFreq::Last)
        LogState(hostGrid.get(), iter);

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    // Free device memory
    hipFree(curDataDevice);
    hipFree(prevDataDevice);

    return (duration.count() * 1e-6);
}

int main(int argc, char** argv)
{
    int width, height;
    float alpha, beta, gamma;

    if (!ReiterSimulation::ParseInputParams(argc, argv, &width, &height, &alpha, &beta, &gamma))
    {
        printf("Correct usage should be: %s <width> <height> <alpha> <beta> <gamma>\n", argv[0]);
        return -1;
    }

    ReiterCUDA model(width, height);
    auto dur = model.RunSimulation(alpha, beta, gamma);

    printf("{type: \"CUDA\", elapsed: %lf, width: %d, height: %d, alpha: %f, beta: %f, gamma: %f},\n", dur, width, height, alpha, beta, gamma);

    return 0;
}
