#include "hip/hip_runtime.h"
#include "ReiterCUDA.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <stdio.h>

__device__ void GetNeighbourCellIds(size_t cellId, size_t* outIdArray, int width)
{
    size_t j = cellId % width;
    size_t i = (cellId - j) / width;

    int nOff;
    if (j%2 == 0)
        nOff = -1;
    else
        nOff = 0;
        
    outIdArray[0] = width * (i-1) + j;
    outIdArray[1] = width * (nOff + i) + j - 1;
    outIdArray[2] = width * (nOff + i+1) + j - 1;
    outIdArray[3] = width * (nOff + i) + j + 1;
    outIdArray[4] = width * (nOff + i+1) + j + 1;
    outIdArray[5] = width * (i+1) + j;
}

__device__ bool CheckReceptiveCell(float* data, size_t cellId, int width, int height)
{
    if(data[cellId] >= 1)
        return true;

    int j = cellId % width;
    int i = (cellId - j) / width;

    int nOff;
    if (j%2 == 0)
        nOff = -1;
    else
        nOff = 0;
        
    if(i>0 && data[width * (i-1) + j] >= 1)
        return true;
    if(j>0 && (nOff + i) > 0 && data[width * (nOff + i) + j - 1] >= 1)
        return true;
    if(j>0 && (nOff + i+1) < height && data[width * (nOff + i+1) + j - 1] >= 1)
        return true;
    if(j+1 < width && (nOff + i) > 0 && data[width * (nOff + i) + j + 1] >= 1)
        return true;
    if(j+1 < width && (nOff + i+1) < height && data[width * (nOff + i+1) + j + 1] >= 1)
        return true;
    if(i+1 < height && data[width * (i+1) + j] >= 1)
        return true;

    return false;
}

__global__ void simulationKernel(float* curData, float* prevData, int height, int width, float alpha, float beta, float gamma)
{
    int cellId = blockIdx.x * blockDim.x + threadIdx.x;

    if (cellId >= height * width)
        return;

    size_t idArray[6];
    
    int j = cellId % width;
    int i = (cellId - j) / width;

    if (i == 0 || j == 0 || height - i == 1 || width - j == 1)
        return;

    GetNeighbourCellIds(cellId, idArray, width);

    float sum = 0;
    for (int k = 0; k < 6; k++) {
        int id = idArray[k];
        if (!CheckReceptiveCell(prevData, id, width, height))
            sum += prevData[id];
    }

    float cellR = (CheckReceptiveCell(prevData, cellId, width, height) ? 1.0 : 0.0);
    float cellU = (cellR == 0.0 ? prevData[cellId] : 0.0);

    curData[cellId] = prevData[cellId] + (alpha / 2.0) * ((sum / 6.0) - cellU) + (gamma * cellR);
    
}

double ReiterCUDA::RunSimulation(float alpha, float beta, float gamma)
{
    // Device data
    float* curDataDevice;
    float* prevDataDevice;

    // Allocate host memory
    auto hostGrid = CreateGrid(beta);

    // Allocate device memory
    hipMalloc((void**)&curDataDevice, m_Height * m_Width * sizeof(float));
    hipMalloc((void**)&prevDataDevice, m_Height * m_Width * sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();

    // Copy initial data to device
    hipMemcpy(curDataDevice, hostGrid.get(), m_Height * m_Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(prevDataDevice, hostGrid.get(), m_Height * m_Width * sizeof(float), hipMemcpyHostToDevice);

    bool stable = false;
    size_t iter = 0;

    LogState(hostGrid.get(), 0);
    while (!stable && iter <= MAX_ITER)
    {
        stable = true;

        int blockSize = 256;
        int gridSize = (m_Height * m_Width + blockSize - 1) / blockSize;
        simulationKernel<<<gridSize, blockSize>>>(curDataDevice, prevDataDevice, m_Height, m_Width, alpha, beta, gamma);

        hipDeviceSynchronize();

        auto tmp = curDataDevice;
        curDataDevice = prevDataDevice;
        prevDataDevice = tmp;

        iter++;
        stable = false;
    }

    // Get data from device
    hipMemcpy(hostGrid.get(), curDataDevice, m_Height * m_Width * sizeof(float), hipMemcpyDeviceToHost);
    LogState(hostGrid.get(), iter);

    auto stop = std::chrono::high_resolution_clock::now();
    printf("Simulation took %ld iterations\n", iter);
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    // Free device memory
    hipFree(curDataDevice);
    hipFree(prevDataDevice);

    return (duration.count() * 1e-6);
}

int main(int argc, char** argv)
{
    ReiterCUDA model(100, 100);
    auto dur = model.RunSimulation(1, 0.2, 0.01);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Ran on %d GPU(s)\n", deviceCount);
    printf("Execution took %lf seconds\n", dur);

    return 0;
}
